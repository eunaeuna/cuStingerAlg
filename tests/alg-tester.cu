#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include "algs.cuh"

#include "static_breadth_first_search/bfs_top_down.cuh"
#include "static_breadth_first_search/bfs_bottom_up.cuh"
#include "static_breadth_first_search/bfs_hybrid.cuh"
#include "static_connected_components/cc.cuh"
#include "static_page_rank/pr.cuh"
#include "static_betweenness_centrality/bc.cuh"

#include "streaming_page_rank/pr.cuh"

using namespace cuStingerAlgs;


#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)

#define PR_UPDATE 1

#if PR_UPDATE
// RNG using Lehmer's Algorithm ================================================
#define RNG_A 16807
#define RNG_M 2147483647
#define RNG_Q 127773
#define RNG_R 2836
#define RNG_SCALE (1.0 / RNG_M)

// Seed can always be changed manually
static int seed = 1;
double getRand(){

    int k = seed / RNG_Q;
    seed = RNG_A * (seed - k * RNG_Q) - k * RNG_R;

    if (seed < 0) {
        seed += RNG_M;
    }

    return seed * (double) RNG_SCALE;
}

void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst){
        printf("-----------------------------------------------\n");
        for(int32_t e=0; e<numEdges;e++){
#if 0 //rand        	    
                edgeSrc[e] = rand()%nv;
                edgeDst[e++] = rand()%nv;
                printf("edgeSrc[%d]=%d,\t edgeDst[%d]=%d\n",e,edgeSrc[e-1],e-1,edgeDst[e-1]);
                edgeSrc[e] = edgeDst[e-1];
                edgeDst[e] = edgeSrc[e-1];
                printf("edgeSrc[%d]=%d,\t edgeDst[%d]=%d\n",e,edgeSrc[e+1],e,edgeDst[e]);
#else
                edgeSrc[e] = 2;
                edgeDst[e++] = 3;
                printf("edgeSrc[%d]=%d,\t edgeDst[%d]=%d\n",e,edgeSrc[e-1],e-1,edgeDst[e-1]);
                edgeSrc[e] = edgeDst[e-1];
                edgeDst[e] = edgeSrc[e-1];
                printf("edgeSrc[%d]=%d,\t edgeDst[%d]=%d\n",e,edgeSrc[e],e,edgeDst[e]);           
#endif                
        }
}
#if 0
void generateEdgeUpdatesRMAT(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst,double A, double B, double C, double D){
        int64_t src,dst;
        int scale = (int)log2(double(nv));
        for(int32_t e=0; e<numEdges; e++){
                rmat_edge(&src,&dst,scale, A,B,C,D);
                edgeSrc[e] = src;
                edgeDst[e] = dst;
        }
}
#endif

void printcuStingerUtility(cuStinger custing, bool allInfo){
        length_t used,allocated;

        used     =custing.getNumberEdgesUsed();
        allocated=custing.getNumberEdgesAllocated();
        if (allInfo)
                cout << ", " << used << ", " << allocated << ", " << (float)used/(float)allocated;
        else
                cout << ", " << (float)used/(float)allocated;
}

#endif


int main(const int argc, char *argv[]){
	int device=0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne,*off;
    vertexId_t *adj;

	bool isDimacs,isSNAP,isRmat=false,isMarket;
	string filename(argv[1]);
	isDimacs = filename.find(".graph")==std::string::npos?false:true;
	isSNAP   = filename.find(".txt")==std::string::npos?false:true;
	isRmat 	 = filename.find("kron")==std::string::npos?false:true;
	isMarket = filename.find(".mtx")==std::string::npos?false:true;

	if(isDimacs){
	    readGraphDIMACS(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isSNAP){
	    readGraphSNAP(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isMarket){
		readGraphMatrixMarket(argv[1],&off,&adj,&nv,&ne,(isRmat)?false:true);
	}
	else{ 
		cout << "Unknown graph type" << endl;
	}

	cout << "Vertices: " << nv << "    Edges: " << ne << endl;

	hipEvent_t ce_start,ce_stop;
	cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

	cuStingerInitConfig hipInit;
	hipInit.initState =eInitStateCSR;
	hipInit.maxNV = nv+1;
	hipInit.useVWeight = false;
	hipInit.isSemantic = false;  // Use edge types and vertex types
	hipInit.useEWeight = false;
	// CSR data
	hipInit.csrNV 			= nv;
	hipInit.csrNE	   		= ne;
	hipInit.csrOff 			= off;
	hipInit.csrAdj 			= adj;
	hipInit.csrVW 			= NULL;
	hipInit.csrEW			= NULL;

	custing.initializeCuStinger(hipInit);

	
	float totalTime;
#if 0
	ccBaseline scc;
	scc.Init(custing);
	scc.Reset();
	start_clock(ce_start, ce_stop);
//	scc.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	// cout << "The number of iterations           : " << scc.GetIterationCount() << endl;
	// cout << "The number of connected-compoents  : " << scc.CountConnectComponents(custing) << endl;
	// cout << "Total time for connected-compoents : " << totalTime << endl; 
	scc.Release();

	ccConcurrent scc2;
	scc2.Init(custing);
	scc2.Reset();
	start_clock(ce_start, ce_stop);
    // scc2.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	// cout << "The number of iterations           : " << scc2.GetIterationCount() << endl;
	// cout << "The number of connected-compoents  : " << scc2.CountConnectComponents(custing) << endl;
	// cout << "Total time for connected-compoents : " << totalTime << endl; 
	scc2.Release();


	ccConcurrentLB scc3;
	scc3.Init(custing);
	scc3.Reset();
	start_clock(ce_start, ce_stop);
	scc3.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations           : " << scc3.GetIterationCount() << endl;
	cout << "The number of connected-compoents  : " << scc3.CountConnectComponents(custing) << endl;
	cout << "Total time for connected-compoents : " << totalTime << endl; 
	scc3.Release();


	// ccConcurrentOptimized scc4;
	// scc4.Init(custing);
	// scc4.Reset();
	// start_clock(ce_start, ce_stop);
	// scc4.Run(custing);
	// totalTime = end_clock(ce_start, ce_stop);
	// cout << "The number of iterations           : " << scc4.GetIterationCount() << endl;
	// cout << "The number of connected-compoents  : " << scc4.CountConnectComponents(custing) << endl;
	// cout << "Total time for connected-compoents : " << totalTime << endl; 
	// scc4.Release();

	// Finding largest vertex

	vertexId_t maxV=0;
	length_t   maxLen=0;
	for(int v=1; v<nv;v++){
		if((off[v+1]-off[v])>maxLen){
			maxV=v;
			maxLen=off[v+1]-off[v];
		}
	}
	// cout << "Largest vertex is: " << maxV << "   With the length of :" << maxLen << endl;

	bfsTD bfs;
	bfs.Init(custing);
	bfs.Reset();
	bfs.setInputParameters(maxV);
	start_clock(ce_start, ce_stop);
	bfs.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);

	cout << "The number of levels          : " << bfs.getLevels() << endl;
	cout << "The number of elements found  : " << bfs.getElementsFound() << endl;
	cout << "Total time for BFS - Top-Down : " << totalTime << endl; 

	bfs.Release();

	bfsBU bfsbu;
	bfsbu.Init(custing);
	bfsbu.Reset();
	bfsbu.setInputParameters(maxV);
	start_clock(ce_start, ce_stop);
	bfsbu.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);

	cout << "The number of levels          : " << bfsbu.getLevels() << endl;
	cout << "The number of elements found  : " << bfsbu.getElementsFound(custing) << endl;
	cout << "Total time for BFS - Bottom-up: " << totalTime << endl; 

	bfsbu.Release();

	bfsHybrid bfsHy;
	bfsHy.Init(custing);
	bfsHy.Reset();
	bfsHy.setInputParameters(maxV);
	start_clock(ce_start, ce_stop);
	bfsHy.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);

	cout << "The number of levels          : " << bfsHy.getLevels() << endl;
	cout << "The number of elements found  : " << bfsHy.getElementsFound(custing) << endl;
	cout << "Total time for BFS - Hybrid   : " << totalTime << endl; 

	bfsHy.Release();
#endif

#if 0
	StaticPageRank pr;
	pr.Init(custing);
	pr.Reset();
	pr.setInputParameters(50,0.000001);
	start_clock(ce_start, ce_stop);
	pr.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations      : " << pr.getIterationCount() << endl;
	cout << "Total time for pagerank       : " << totalTime << endl; 
	cout << "Average time per iteartion "
			"   : " << totalTime/(float)pr.getIterationCount() << endl; 
	pr.printRankings(custing);

	pr.Release();
#endif

    StreamingPageRank upr;
    upr.Init(custing);
    upr.Reset();
    upr.setInputParameters(50,0.00001);

    start_clock(ce_start, ce_stop);
    upr.Run(custing);
    totalTime = end_clock(ce_start, ce_stop);
    cout << "=============================================" << endl;
    cout << "The number of iterations      : " << upr.getIterationCount() << endl;
    cout << "Total time for streaming pagerank       : " << totalTime << endl;
    cout << "Average time per iteartion    : " << totalTime/(float)upr.getIterationCount() << endl;
    upr.printRankings(custing);
    
#if 1
	//------------------------
	// update
	//------------------------
	        //TO DO: adding termination conditions; iterations and epsilon
	        //upr.setInputParameters(30,0.0000001);  //initialize hostRPData

	        //graph update
	        length_t *len = (length_t *)malloc(sizeof(length_t)*(nv));
	        for(unsigned i=0; i<nv; ++i){
	           len[i] = off[i+1] - off[i];
	        }

	        int numBatches = 1;
	        std::vector<BatchUpdateData*>buds(numBatches);
	        int numEdges = 1;
	        length_t numTotalEdges = numEdges;

	        for(unsigned i=0; i<numBatches; ++i){
	            buds[i] = new BatchUpdateData(numTotalEdges*2,true,nv); //undirected
	        	//buds[i] = new BatchUpdateData(numTotalEdges,true,nv);
	        }

	        for(unsigned i=0; i<numBatches; ++i){
	           BatchUpdateData& bud = *buds[i];

	            if(isRmat){
	            //  double a = 0.55, b = 0.15, c = 0.15,d = 0.25;
	            //  generateEdgeUpdatesRMAT(nv, numEdges, bud.getSrc(),bud.getDst(),a,b,c,d);
	            }
	            else{
	                generateEdgeUpdates(nv, numEdges*2, bud.getSrc(),bud.getDst());
	            }
	        }

	        length_t *newOff = (length_t *)malloc(sizeof(length_t)*(nv+1));
	        length_t sum = 0;
	        for(unsigned i=0; i<nv+1; ++i){
	           newOff[i] = sum;
	           sum += len[i];
	        }
	        vertexId_t *newAdj = (vertexId_t*)malloc(sizeof(vertexId_t)*(newOff[nv]));
	        //populate newAdj
	        for(unsigned i=0, j=0; i<ne; ++i) {
	           if(adj[i] != -1) newAdj[j++] = adj[i];
	        }

	        hipInit.csrNE = newOff[nv];
	        hipInit.csrOff = newOff;
	        hipInit.csrAdj = newAdj;

	        cuStinger custingTest(defaultInitAllocater,defaultUpdateAllocater);
	        custingTest.initializeCuStinger(hipInit);

	        int sps = 128; //block size
	        length_t allocs;
	        BatchUpdate bu1(*buds[0]);
	        bu1.sortDeviceBUD(sps);
	        custingTest.edgeInsertionsSorted(bu1, allocs);

	        printcuStingerUtility(custingTest, false);

	        start_clock(ce_start, ce_stop);

#define SPR_ON 1
#if SPR_ON //streaming pr
	        printf("\n<spr>======================================\n");
	        //upr.UpdateDiff(custing, *buds[0]);
	        upr.UpdateDiff(custingTest, *buds[0]);
	        //upr.setInputParameters(50,0.00001);
	        //upr.Run2(custingTest);
#else
	        printf("\n<pr>======================================\n");
	        upr.setInputParameters(1,0.00001);
	        upr.Run(custingTest);
#endif	        
	        totalTime = end_clock(ce_start, ce_stop);	        
	        cout << "The number of iterations      : " << upr.getIterationCount() << endl;
	        cout << "Total time for updating streaming pagerank       : " << totalTime << endl;
	        cout << "Average time per iteartion    : " << totalTime/(float)upr.getIterationCount() << endl;

	        //upr.printRankings(custingTest);	        
#if SPR_ON //streaming pr	        
	        upr.printRankings(custingTest);
#else
	        upr.printRankings(custingTest);
#endif	
#endif //end of update   

	custing.freecuStinger();

	free(off);
	free(adj);
    return 0;	
}

